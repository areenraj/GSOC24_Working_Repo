#include <cstdlib> 
#include <iostream> 
#include <fstream>
#include <chrono>
#include "cblas.h"
#include <time.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

int main()
{

    /*Define Initial Dimension (M), Final Dimenions (limit), and incremental dimension every iteration (inc)*/
    int limit = 10000;
    int inc = 500;
    int M = 5;
    
    /*Create output file*/
    std::ofstream parallel("parallel.txt");

    parallel.open("parallel.txt", std::ios::app);
    parallel << "Dimension" << "\t\t\t" << "Time(ms)" << "\n";
    parallel.close();
   
    float alpha = 1.0;
    float beta = 0.0;
    /*While Loop that increases the dimension of the square matrices each iteration*/

    while(M<limit)
    {

        hipblasHandle_t handle; 

        double time = 0.0;

        /*Allocate Memory to input square matrices of dimensions MxM (a and b) and output matrix mul*/
        float *a = (float*) malloc(sizeof(float)*M*M);
        float *b = (float*) malloc(sizeof(float)*M*M);
        float *mul = (float*) malloc(sizeof(float)*M*M);

        float  *devPtra, *devPtrb, *devPtrmul;
    
        hipMalloc ((void**)&devPtra, M*M*sizeof(float));
        hipMalloc ((void**)&devPtrb, M*M*sizeof(float));
        hipMalloc ((void**)&devPtrmul, M*M*sizeof(float));

        /*initialize the input matrices to random values and the output matrix to zero*/
        for(int i=0;i<M;i++)    
        {    
            for(int j=0;j<M;j++)    
            {    
                a[IDX2C(i,j,M)] = 2500;
                b[IDX2C(i,j,M)] = 2500;
                mul[IDX2C(i,j,M)] = 0.0;
            }    
        }    
        
        /*OpenBLAS computations*/
        
        hipblasCreate(&handle);
        hipblasSetMatrix (M, M, sizeof(*a), a, M, devPtra, M);
        hipblasSetMatrix (M, M, sizeof(*b), b, M, devPtrb, M);
        hipblasSetMatrix (M, M, sizeof(*mul), b, M, devPtrmul, M);

        for(int i=0; i<200; i++)
        {

            auto start = std::chrono::high_resolution_clock::now();
            hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, M, M, &alpha , devPtra, M, devPtrb, M, &beta, devPtrmul, M);
            hipDeviceSynchronize();
            auto stop = std::chrono::high_resolution_clock::now();
            auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);    

            time += duration.count(); 

        
        }
 
        time = time/200.0;

        /*Write duration to file along with dimension number*/
        parallel.open("parallel.txt", std::ios::app);
        parallel << M << "\t\t\t" << time << "\n";
        parallel.close();

        free(a);
        free(b);
        free(mul);
        hipFree(devPtra);
        hipFree(devPtrb);
        hipFree(devPtrmul);

        hipblasDestroy(handle);

        M += inc;

    } 


    return 0;

}
